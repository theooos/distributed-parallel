#include "hip/hip_runtime.h"
/*
 * Theodore Gregory - 1453831
 *
 * Tasks:
 * Block scan
 * Full scan for large vectors
 * Bank Conflict Avoidance Optimisation
 *
 * Timings:
 * Block scan without BCAO
 * Block scan with BCAO
 * Full scan without BCAO
 * Full scan with BCAO
 *
 * Hardware:
 * CPU - Intel - Core i5-6600 3.3GHz Quad-Core Processor
 * GPU - Zotac - GeForce GTX 1080 8GB AMP! Edition Video Card
 *
 * Implementation details:
 * TODO Any details or performance strategies I implemented which improve upon a base level of the target goals
 */

#include <stdio.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>
#include <helper_functions.h>

// A helper macro to simplify handling cuda error checking
#define CUDA_ERROR( err, msg ) { \
if (err != hipSuccess) {\
    printf( "%s: %s in %s at line %d\n", msg, hipGetErrorString( err ), __FILE__, __LINE__);\
    exit( EXIT_FAILURE );\
}\
}

#define BLOCK_SIZE 1024

__global__ void
prescan(int *g_odata, int *g_idata, int n)
{
	extern __shared__ int temp[];  // allocated on invocation
	int thid = threadIdx.x;
	int offset = 1;
	temp[2*thid] = g_idata[2*thid]; // load input into shared memory
	temp[2*thid+1] = g_idata[2*thid+1];
	for (int d = n>>1; d > 0; d >>= 1)                    // build sum in place up the tree
	{
		__syncthreads();
		if (thid < d)
		{
			int ai = offset*(2*thid+1)-1;
			int bi = offset*(2*thid+2)-1;
			temp[bi] += temp[ai];
		}
		offset *= 2;
	}
	if (thid == 0) { temp[n - 1] = 0; } // clear the last element
	for (int d = 1; d < n; d *= 2) // traverse down tree & build scan
	{
		offset >>= 1;
		__syncthreads();
		if (thid < d)
		{
			int ai = offset*(2*thid+1)-1;
			int bi = offset*(2*thid+2)-1;
			int t = temp[ai];
			temp[ai] = temp[bi];
			temp[bi] += t;
		}
	}
	__syncthreads();
	g_odata[2*thid] = temp[2*thid]; // write results to device memory
	g_odata[2*thid+1] = temp[2*thid+1];
}


int main(void)
{
	hipError_t err = hipSuccess;

	// ***************** Initial variable construction ******************
	hipEvent_t start, stop;
	float time_bscan, time_bscan_bcao, time_fscan, time_fscan_bcao;
	hipEventCreate(&start);
	hipEventCreate(&stop);

	uint num_elements = 10000000;
	size_t size = num_elements * sizeof(float);
	int grid_size = 1 + (num_elements - 1) / BLOCK_SIZE;

	// Allocate the input and output vector
	int *h_input_array = (int *)malloc(size);
	int *h_gpu_results = (int *)malloc(size);
	int *h_host_results = (int *)malloc(size);

	// Verify that allocations succeeded
	if(h_input_array == NULL || h_host_results == NULL){
		fprintf(stderr, "Failed to allocate host vectors!\n");
		exit(EXIT_FAILURE);
	}

	// Initialise the host input and output vectors
	for(int i = 0; i < num_elements; i++){
		h_input_array[i] = rand()%10;
	}
	h_host_results[0] = 0;
	for(int i = 1; i < num_elements; i++){
		h_host_results[i] = h_host_results[i-1] + h_input_array[i-1];
	}

	// Check host vectors are as expected
	printf("%d %d %d %d\n", h_input_array[0], h_input_array[1], h_input_array[2], h_input_array[num_elements-1]);
	printf("%d %d %d %d\n", h_host_results[0], h_host_results[1], h_host_results[2], h_host_results[num_elements-1]);

	// *************************** BSCAN **********************************



	// *************************** BSCAN BCAO *****************************



	// *************************** FSCAN **********************************



	// *************************** FSCAN BCAO******************************




	// Clean up the Device timer event objects
	hipEventDestroy(start);
	hipEventDestroy(stop);

	// Reset the device and exit
	err = hipDeviceReset();
	CUDA_ERROR(err, "Failed to reset the device");
	return 0;
}
